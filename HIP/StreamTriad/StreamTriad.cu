#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
extern "C" {
   #include "timer.h"
}

#define NTIMES 16

// CUDA kernel version of stream triad
__global__ void StreamTriad(int n, double scalar, double *a, double *b, double *c){
   int i = blockIdx.x*blockDim.x+threadIdx.x;

   // Protect from going out-of-bounds
   if (i >= n) return;

   c[i] = a[i] + scalar*b[i];
}

int main(int argc, char *argv[]){
   struct timespec tkernel, ttotal;
   // initializing data and arrays
   int stream_array_size = 80000000;
   double scalar = 3.0, tkernel_sum = 0.0, ttotal_sum = 0.0;

   // allocate host memory and initialize
   double *a = (double *)malloc(stream_array_size*sizeof(double));
   double *b = (double *)malloc(stream_array_size*sizeof(double));
   double *c = (double *)malloc(stream_array_size*sizeof(double));

   for (int i=0; i<stream_array_size; i++) {
      a[i] = 1.0;
      b[i] = 2.0;
   }

   // allocate device memory. suffix of _d indicates a device pointer
   double *a_d, *b_d, *c_d;
   hipMalloc(&a_d, stream_array_size*sizeof(double));
   hipMalloc(&b_d, stream_array_size*sizeof(double));
   hipMalloc(&c_d, stream_array_size*sizeof(double));

   // setting block size and padding total grid size to get even block sizes
   int blocksize = 512;
   int gridsize = (stream_array_size + blocksize - 1)/blocksize;

   for (int k=0; k<NTIMES; k++){
      cpu_timer_start(&ttotal);
      // copying array data from host to device
      hipMemcpy(a_d, a, stream_array_size*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(b_d, b, stream_array_size*sizeof(double), hipMemcpyHostToDevice);
      // cuda memcopy to device returns after buffer available, so synchronize to
      // get accurate timing for kernel only
      hipDeviceSynchronize();

      cpu_timer_start(&tkernel);
      // launch stream triad kernel
      StreamTriad<<<gridsize, blocksize>>>(stream_array_size, scalar, a_d, b_d, c_d);
      // need to force completion to get timing
      hipDeviceSynchronize();
      tkernel_sum += cpu_timer_stop(tkernel);

      // cuda memcpy from device to host blocks for completion so no need for synchronize
      hipMemcpy(c, c_d, stream_array_size*sizeof(double), hipMemcpyDeviceToHost);
      ttotal_sum += cpu_timer_stop(ttotal);
      // check results and print errors if found. limit to only 10 errors per iteration
      for (int i=0, icount=0; i<stream_array_size && icount < 10; i++){
         if (c[i] != 1.0 + 3.0*2.0) {
            printf("Error with result c[%d]=%lf on iter %d\n",i,c[i],k);
            icount++;
         }
      }
   }
   printf("Average runtime is %lf msecs data transfer is %lf msecs\n",
           tkernel_sum/NTIMES, (ttotal_sum - tkernel_sum)/NTIMES);

   hipFree(a_d);
   hipFree(b_d);
   hipFree(c_d);

   free(a);
   free(b);
   free(c);
}
