#include "hip/hip_runtime.h"
#define MIN_REDUCE_SYNC_SIZE warpSize

__device__ void reduction_sum_within_block(double  *spad)
{
   const unsigned int tiX  = threadIdx.x;
   const unsigned int ntX  = blockDim.x;

   for (int offset = ntX >> 1; offset > MIN_REDUCE_SYNC_SIZE; offset >>= 1) {
      if (tiX < offset) {
         spad[tiX] = spad[tiX] + spad[tiX+offset];
      }
      __syncthreads();
   }
   if (tiX < MIN_REDUCE_SYNC_SIZE) {
      for (int offset = MIN_REDUCE_SYNC_SIZE; offset > 1; offset >>= 1) {
         spad[tiX] = spad[tiX] + spad[tiX+offset];
         __syncthreads();
      }
      spad[tiX] = spad[tiX] + spad[tiX+1];
   }
}

__global__ void reduce_sum_stage1of2(
                 const int      isize,      // 0  Total number of cells.
                 const double  *array,      // 1
                       double  *blocksum,   // 2
                       double  *redscratch) // 3
{
    extern __shared__ double spad[];
    const unsigned int giX  = blockIdx.x*blockDim.x+threadIdx.x;
    const unsigned int tiX  = threadIdx.x;

    const unsigned int group_id = blockIdx.x;

    spad[tiX] = 0.0;
    if (giX < isize) {
      spad[tiX] = array[giX];
    }

    __syncthreads();

    reduction_sum_within_block(spad);

    //  Write the local value back to an array size of the number of groups
    if (tiX == 0){
      redscratch[group_id] = spad[0];
      (*blocksum) = spad[0];
    }
}

__global__ void reduce_sum_stage2of2(
                 const int    isize,
                       double *total_sum,
                       double *redscratch)
{
   extern __shared__ double spad[];
   const unsigned int tiX  = threadIdx.x;
   const unsigned int ntX  = blockDim.x;

   int giX = tiX;

   spad[tiX] = 0.0;

   // load the sum from reduction scratch, redscratch
   if (tiX < isize) spad[tiX] = redscratch[giX];

   for (giX += ntX; giX < isize; giX += ntX) {
      spad[tiX] += redscratch[giX];
   }

   __syncthreads();

   reduction_sum_within_block(spad);

   if (tiX == 0) {
     (*total_sum) = spad[0];
   }
}

#include <stdio.h>
#include <sys/time.h>
extern "C" {
   #include "timer.h"
}

int main(int argc, char *argv[]){

   size_t nsize = 200;

   double *x = (double *)malloc(nsize*sizeof(double));

   for (int i = 0; i<nsize; i++){
     //x[i] = rand()*100.0;
     x[i] = (double)i;
   }

   struct timespec tstart_cpu;
   cpu_timer_start(&tstart_cpu);

   size_t blocksize = 128;
   size_t blocksizebytes = blocksize*sizeof(double);
   size_t global_work_size = ((nsize + blocksize - 1) /blocksize) * blocksize;
   size_t gridsize     = global_work_size/blocksize;

   double *dev_x, *dev_total_sum, *dev_redscratch;
   hipMalloc(&dev_x, nsize*sizeof(double));
   hipMalloc(&dev_total_sum, 1*sizeof(double));
   hipMalloc(&dev_redscratch, gridsize*sizeof(double));

   hipMemcpy(dev_x, x, nsize*sizeof(double), hipMemcpyHostToDevice);

   reduce_sum_stage1of2<<<gridsize, blocksize, blocksizebytes>>>(nsize, dev_x, dev_total_sum, dev_redscratch);

   if (gridsize > 1) {
      reduce_sum_stage2of2<<<1, blocksize, blocksizebytes>>>(nsize, dev_total_sum, dev_redscratch);
   }

   double total_sum;
   hipMemcpy(&total_sum, dev_total_sum, 1*sizeof(double), hipMemcpyDeviceToHost);
   printf("Result -- total sum %lf \n",total_sum);

   hipFree(dev_redscratch);
   hipFree(dev_total_sum);
   hipFree(dev_x);

   free(x);
}
