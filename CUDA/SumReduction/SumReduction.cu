#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
extern "C" {
   #include "timer.h"
}

#define MIN_REDUCE_SYNC_SIZE 32

#define REDUCE_IN_TILE(operation, _spad_arr)                                    \
    for (int offset = ntX >> 1; offset > MIN_REDUCE_SYNC_SIZE; offset >>= 1)    \
    {                                                                           \
        if (tiX < offset)                                                       \
        {                                                                       \
            _spad_arr[tiX] = operation(_spad_arr[tiX], _spad_arr[tiX+offset]);  \
        }                                                                       \
        __syncthreads();                                                        \
    }                                                                           \
    if (tiX < MIN_REDUCE_SYNC_SIZE)                                             \
    {                                                                           \
        for (int offset = MIN_REDUCE_SYNC_SIZE; offset > 1; offset >>= 1)       \
        {                                                                       \
            _spad_arr[tiX] = operation(_spad_arr[tiX], _spad_arr[tiX+offset]);  \
            __syncthreads();                                                    \
        }                                                                       \
        _spad_arr[tiX] = operation(_spad_arr[tiX], _spad_arr[tiX+1]);           \
    }

__device__ double SUM(double a, double b)
{
    return a + b; 
}

__device__ void reduction_sum_within_spad(double  *spad)
{
   const unsigned int tiX  = threadIdx.x;
   const unsigned int ntX  = blockDim.x;

   REDUCE_IN_TILE(SUM, spad);
}

__global__ void reduce_sum_stage1of2(
                 const int      isize,      // 0  Total number of cells.
                       double  *array,      // 1 
                       double  *blocksum,   // 2 
                       double  *redscratch) // 3
{
    extern __shared__ double spad[];
    const unsigned int giX  = blockIdx.x*blockDim.x+threadIdx.x;
    const unsigned int tiX  = threadIdx.x;

    const unsigned int group_id = blockIdx.x;

    spad[tiX] = 0.0;
    if (giX < isize) {
      spad[tiX] = array[giX];
    }    

    __syncthreads();

    reduction_sum_within_spad(spad);

    //  Write the local value back to an array size of the number of groups
    if (tiX == 0){
      redscratch[group_id] = spad[0];
      (*blocksum) = spad[0];
    }    
}

__global__ void reduce_sum_stage2of2(
                 const int    isize,
                       double *total_sum,
                       double *redscratch)
{
   extern __shared__ double spad[];
   const unsigned int tiX  = threadIdx.x;
   const unsigned int ntX  = blockDim.x;

   int giX = tiX; 

   spad[tiX] = 0.0; 

   // load the sum from reduction scratch, redscratch
   if (tiX < isize) spad[tiX] = redscratch[giX];

   for (giX += ntX; giX < isize; giX += ntX) {
      spad[tiX] += redscratch[giX];
   }

   __syncthreads();

   reduction_sum_within_spad(spad);

   if (tiX == 0) { 
     (*total_sum) = spad[0];
   }
}


int main(int argc, char *argv[]){

   size_t nsize = 200;

   double *x = (double *)malloc(nsize*sizeof(double));

   for (int i = 0; i<nsize; i++){
     //x[i] = rand()*100.0;
     x[i] = 1.0;
   }   

   struct timespec tstart_cpu;
   cpu_timer_start(&tstart_cpu);

   size_t blocksize = 128; 
   size_t blocksizebytes = blocksize*sizeof(double); 
   size_t global_work_size = ((nsize + blocksize - 1) /blocksize) * blocksize;
   size_t gridsize     = global_work_size/blocksize;

   double *dev_x, *dev_total_sum, *dev_redscratch;
   hipMalloc(&dev_x, nsize*sizeof(double));
   hipMalloc(&dev_total_sum, 1*sizeof(double));
   hipMalloc(&dev_redscratch, gridsize*sizeof(double));

   hipMemcpy(dev_x, x, nsize*sizeof(double), hipMemcpyHostToDevice);

   reduce_sum_stage1of2<<<gridsize, blocksize, blocksizebytes>>>(nsize, dev_x, dev_total_sum, dev_redscratch);

   if (gridsize > 1) {
      reduce_sum_stage2of2<<<gridsize, blocksize, blocksizebytes>>>(nsize, dev_total_sum, dev_redscratch);
   }

   double total_sum;
   hipMemcpy(&total_sum, dev_total_sum, 1*sizeof(double), hipMemcpyDeviceToHost);
   printf("Result -- total sum %lf \n",total_sum);

   hipFree(dev_total_sum);
   hipFree(dev_x);

   free(x);
}
